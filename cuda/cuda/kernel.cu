
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include "omp.h"

#define N 21
#define THREADS 1

__global__ void prime_kernel(int *prime_count) {
  int threads = blockDim.x,
    blocks = gridDim.x,
    thread_id = threadIdx.x,
    block_id = blockIdx.x;
  int number, i, j, k, count;
  bool prime;
  for (i = block_id; i < N; i += blocks) {
    count = 0;
    number = pow((double)2, (double)i);
    for (j = thread_id; j <= number; j += threads) {
      if (j % 2 == 0) {
        count += j == 2;
        continue;
      }
      prime = true;
      for (k = 3; k * k <= j; k += 2) {
        if (j % k == 0) {
          prime = false;
          break;
        }
      }
      if (prime && j > 1) count += 1;
    }
    atomicAdd(&prime_count[i], count);
  }
}

int main() {
  int *prime_count, *prime_count_gpu;
  double t1, t2;

  t1 = omp_get_wtime();
  prime_count = (int*)calloc(sizeof(int), N);
  t2 = omp_get_wtime();
  printf("Done allocating (CPU) in %lf s\n", t2 - t1);

  t1 = omp_get_wtime();
  hipMalloc((void**)&prime_count_gpu, sizeof(int) * N);
  t2 = omp_get_wtime();
  printf("Done allocating (GPU) in %lf s\n", t2 - t1);

  t1 = omp_get_wtime();
  hipMemcpy(prime_count_gpu, prime_count, sizeof(int) * N, hipMemcpyHostToDevice);
  t2 = omp_get_wtime();
  printf("Done copying to GPU in %lf s\n", t2 - t1);

  t1 = omp_get_wtime();
  prime_kernel <<<N, THREADS>>> (prime_count_gpu);
  hipDeviceSynchronize();
  t2 = omp_get_wtime();
  printf("Done calculating in %lf s\n", t2 - t1);

  t1 = omp_get_wtime();
  hipMemcpy(prime_count, prime_count_gpu, sizeof(int) * N, hipMemcpyDeviceToHost);
  t2 = omp_get_wtime();
  
  printf("Done copying to CPU in %lf s\n", t2 - t1);
  int i;
  for (i = 0; i < N; i++) {
    printf("%i: %i\n", (int)pow(2, i), prime_count[i]);
  }
  free(prime_count);
  hipFree(prime_count_gpu);
  return 0;
}
